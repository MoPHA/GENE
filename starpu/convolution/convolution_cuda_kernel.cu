#include "hip/hip_runtime.h"
#include <starpu.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include "convolution_core.h"

  template <typename T>
__global__ void convolution_kernel(
    const T *d_f, // Padded matrix
    const unsigned int paddedN, // N+r
    const unsigned int paddedM, // M+r
    const T *d_g, // kernel
    const int r,  // radius
    T *d_h,       // output
    const unsigned int N, // N
    const unsigned int M  // M
    ) 
{
  // Set the padding size and filter size
  unsigned int paddingSize = r;
  unsigned int filterSize = 2 * r + 1;

  // Set the pixel coordinate. 
  // Threads in the padding size wont do anything.

  const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
  const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

  // The multiply-add operation for the pixel coordinate ( j, i )
  if( j >= paddingSize && j < paddedN - paddingSize && i >= paddingSize && i < paddedM - paddingSize )
  {
    unsigned int oPixelPos = ( i - paddingSize ) * N + ( j - paddingSize );

    d_h[oPixelPos] = 0.0;

    for( int k = -r; k <=r; k++ )
    {
      for( int l = -r; l <= r; l++ )
      {
        unsigned int iPixelPos = ( i + k ) * paddedN + ( j + l );
        unsigned int coefPos = ( k + r ) * filterSize + ( l + r );
        d_h[oPixelPos] += d_f[iPixelPos] * d_g[coefPos];
      }
    }

  }
}

inline unsigned int iDivUp( const unsigned int &a, const unsigned int &b )
{
  return ( a%b != 0 ) ? (a/b+1):(a/b);
}

#if 0
extern "C++" void compute_convolution_gpu(
    vector< TestFunction<float>* > &subi, 
    TestKernel<float> &g,
    vector< TestFunction<float>* > &subo, 
    int M, int N
    )
{
  int r = g.radius;
  printf("[GPU]: Compute convolution ... \n");

  // Allocate the memory on a device (corresponding to a smaller conv_matrix)
  // ---------------------------------------------------------------------------- 
  float *d_f = NULL;
  unsigned int paddedMatrixSizeByte = subi[0]->get_mem_size();
  hipMalloc( reinterpret_cast<void **>(&d_f), paddedMatrixSizeByte );

  float *d_h = NULL; 
  unsigned int imageSizeByte = subo[0]->get_mem_size();
  hipMalloc( reinterpret_cast<void **>(&d_h), imageSizeByte );

  float *d_g = NULL;
  unsigned int filterKernelSizeByte = g.get_mem_size();
  hipMalloc( reinterpret_cast<void **>(&d_g), filterKernelSizeByte );

  float *h_g = g.data;        // Kernel
  hipMemcpy( d_g, h_g, filterKernelSizeByte, hipMemcpyHostToDevice ); // Host to Device

  // Setting the execution configuration
  // ---------------------------------------------------------------------------- 
  const unsigned int blockN = 32;
  const unsigned int blockM = 32;
  const dim3 grid( iDivUp( N, blockN ), iDivUp( M, blockM ) );
  const dim3 threadBlock( blockN, blockM );

  printf("Convolution GPU tasks ...\n");
  for (int i=0; i<subi.size(); i++)
  {
    float *h_f = subi[i]->data; // Input
    float *h_h = subo[i]->data; // Output

    // Transfer  from a host to a device
    hipMemcpy(d_f, h_f, paddedMatrixSizeByte, hipMemcpyHostToDevice ); // Host to Device

    // Convolve: call cuda kernel
    convolution_kernel<<<grid,threadBlock>>>(
        d_f, subi[i]->x_num, subi[i]->y_num, 
        d_g, r, 
        d_h, subo[i]->x_num, subo[i]->y_num); 

    hipDeviceSynchronize();

    // Transfer result from the device to the host
    hipMemcpy( h_h, d_h, imageSizeByte, hipMemcpyDeviceToHost ); // Device to Host
  }
}
#endif

extern "C++" void compute_convolution_gpu_func(void *buffers[], void *cl_arg)
{
  float *fo, *fi, *fk;
  size_t no, mo, ni, mi, nk;

  int M, N;
  starpu_codelet_unpack_args(cl_arg, &M, &N);

  // These are cuda pointers
  fo = (float*)STARPU_MATRIX_GET_PTR(buffers[0]);
  no = (unsigned)STARPU_MATRIX_GET_NX(buffers[0]);
  mo = (unsigned)STARPU_MATRIX_GET_NY(buffers[0]);

  fi = (float*)STARPU_MATRIX_GET_PTR(buffers[1]);
  ni = (unsigned)STARPU_MATRIX_GET_NX(buffers[1]);
  mi = (unsigned)STARPU_MATRIX_GET_NY(buffers[1]);

  fk = (float*)STARPU_MATRIX_GET_PTR(buffers[2]);
  nk = (unsigned)STARPU_MATRIX_GET_NX(buffers[2]);

  int r = (nk-1)/2;
  /* printf("%d, %d,%d, %d, %d, %d, %d, %d\n", M, N, mi, ni,  mo, no, nk, r) ; */

#if 0
  const unsigned int blockN = 32;
  const unsigned int blockM = 32;
  const dim3 grid( iDivUp( N, blockN ), iDivUp( M, blockM ) );
  const dim3 threadBlock( blockN, blockM );

  // Convolve: call cuda kernel
  convolution_kernel<<<grid,threadBlock, starpu_cuda_get_local_stream()>>>(fi, ni, mi, fk, r, fo, no, mo);
  hipDeviceSynchronize();
#endif

  const unsigned int blockN = 32;
  const unsigned int blockM = 32;
  const dim3 grid( iDivUp( N, blockN ), iDivUp( M, blockM ) );
  const dim3 threadBlock( blockN, blockM );
  // No shared memory: third parameter is 0
  convolution_kernel<<< grid, threadBlock, 0, starpu_cuda_get_local_stream()>>>(fi, ni, mi, fk, r, fo, no, mo);
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}


